#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <>

#include <string>
#include <torch/extension.h>

#include "sparse_kernels.h"
#include "sparse_layout.cuh"


struct __align__(16) half8 { half2 x, y, z, w; };


/**
 * Compute half-precision sparse matrix multiplication with SDD mode.
 * 
 * It multiplies a dense matrix with another dense matrix and create a new
 * sparse matrix from the sparse layout.
 * 
 * Blocks               : (Sparse Blocks, Total Batches)
 * Threads per Block    : 256
 */
template <bool tr_a, bool tr_b>
__global__ void sparse_hmm_sdd_32x32x32_kernel(
    const half* __restrict__ matrix_a,
    const half* __restrict__ matrix_b,
          half* __restrict__ matrix_c,
    sparse_layout layout, int num_blocks, int size_m, int size_n, int size_k
) {
    __shared__ half2 shared_a[32 * 16 + 32], shared_b[32 * 16 + 32];
    half8 buffer_a, buffer_b, neighbor;
    half2 accum[4][2] = {{{ 0, 0 }}};

    // Load current block and get corresponding row and column positions.
    auto block = layout.get(blockIdx.x);
    int m = block.row() * 32;
    int n = block.col() * 32;

    // Get an offset of each matrix and calculate mapping indices.
    int offset_a = blockIdx.y * size_m * size_k;
    int offset_b = blockIdx.y * size_k * size_n;
    int offset_c = (blockIdx.y * num_blocks + block.idx()) * 32 * 32;

    int p = threadIdx.x / 4;
    int q = threadIdx.x % 4 * 8;
    int r = threadIdx.x / 16 * 4;
    int s = threadIdx.x % 16 * 2;

    // Prefetch first tiles from matrices in global memory.
    buffer_a = *(half8 *) &matrix_a[0];//offset_a + (tr_a ? ((0 + p) * size_m + (m + q)) : ((m + p) * size_k + (0 + q)))];
    buffer_b = *(half8 *) &matrix_b[0];//offset_b + (tr_b ? ((n + p) * size_k + (0 + q)) : ((0 + p) * size_n + (n + q)))];

    #pragma unroll 1
    for (int k = 32; k <= size_k; k += 32) {
        if (tr_a) {
            neighbor.x = __shfl_xor_sync(0xffffffff, buffer_a.x, 16, warpSize);
            neighbor.y = __shfl_xor_sync(0xffffffff, buffer_a.y, 16, warpSize);
            neighbor.z = __shfl_xor_sync(0xffffffff, buffer_a.z, 16, warpSize);
            neighbor.w = __shfl_xor_sync(0xffffffff, buffer_a.w, 16, warpSize);

            buffer_a.x = (p % 2 == 0) ? __lows2half2(buffer_a.x, neighbor.x) : __highs2half2(neighbor.x, buffer_a.x);
            buffer_a.y = (p % 2 == 0) ? __lows2half2(buffer_a.y, neighbor.y) : __highs2half2(neighbor.y, buffer_a.y);
            buffer_a.z = (p % 2 == 0) ? __lows2half2(buffer_a.z, neighbor.z) : __highs2half2(neighbor.z, buffer_a.z);
            buffer_a.w = (p % 2 == 0) ? __lows2half2(buffer_a.w, neighbor.w) : __highs2half2(neighbor.w, buffer_a.w);
        }

        if (!tr_b) {
            neighbor.x = __shfl_xor_sync(0xffffffff, buffer_b.x, 16, warpSize);
            neighbor.y = __shfl_xor_sync(0xffffffff, buffer_b.y, 16, warpSize);
            neighbor.z = __shfl_xor_sync(0xffffffff, buffer_b.z, 16, warpSize);
            neighbor.w = __shfl_xor_sync(0xffffffff, buffer_b.w, 16, warpSize);

            buffer_b.x = (p % 2 == 0) ? __lows2half2(buffer_b.x, neighbor.x) : __highs2half2(neighbor.x, buffer_b.x);
            buffer_b.y = (p % 2 == 0) ? __lows2half2(buffer_b.y, neighbor.y) : __highs2half2(neighbor.y, buffer_b.y);
            buffer_b.z = (p % 2 == 0) ? __lows2half2(buffer_b.z, neighbor.z) : __highs2half2(neighbor.z, buffer_b.z);
            buffer_b.w = (p % 2 == 0) ? __lows2half2(buffer_b.w, neighbor.w) : __highs2half2(neighbor.w, buffer_b.w);
        }

        // Commit the prefetched tiles to the shared memory storage.
        __syncthreads();
        shared_a[tr_a ? ((q + p % 2 + 0) * 16 + (p + q + 0) / 2) : (p * 16 + (0 + q + 0) / 2)] = buffer_a.x;
        shared_a[tr_a ? ((q + p % 2 + 2) * 16 + (p + q + 2) / 2) : (p * 16 + (0 + q + 2) / 2)] = buffer_a.y;
        shared_a[tr_a ? ((q + p % 2 + 4) * 16 + (p + q + 4) / 2) : (p * 16 + (0 + q + 4) / 2)] = buffer_a.z;
        shared_a[tr_a ? ((q + p % 2 + 6) * 16 + (p + q + 6) / 2) : (p * 16 + (0 + q + 6) / 2)] = buffer_a.w;
        shared_b[tr_b ? (p * 16 + (0 + q + 0) / 2) : ((q + p % 2 + 0) * 16 + (p + q + 0) / 2)] = buffer_b.x;
        shared_b[tr_b ? (p * 16 + (0 + q + 2) / 2) : ((q + p % 2 + 2) * 16 + (p + q + 2) / 2)] = buffer_b.y;
        shared_a[tr_b ? (p * 16 + (0 + q + 4) / 2) : ((q + p % 2 + 4) * 16 + (p + q + 4) / 2)] = buffer_b.z;
        shared_a[tr_b ? (p * 16 + (0 + q + 6) / 2) : ((q + p % 2 + 6) * 16 + (p + q + 6) / 2)] = buffer_b.w;
        __syncthreads();

        // Prefetch next tiles from matrices in global memory.
        if (k < size_k) {
            buffer_a = *(half8 *) &matrix_a[0];//offset_a + (tr_a ? ((k + p) * size_m + (m + q)) : ((m + p) * size_k + (k + q)))];
            buffer_b = *(half8 *) &matrix_b[0];//offset_b + (tr_b ? ((n + p) * size_k + (k + q)) : ((k + p) * size_n + (n + q)))];
        }

        // Accumulate the tiled matrix multiplications by loading sliced vectors
        // from the shared memory to local register file.
        #pragma unroll
        for (int i = 0; i < 16; ++ i) {
            half2 reg_a[4], reg_b[2];

            reg_a[0] = shared_a[(r + 0) * 16 + i + (r / 2 + 0) * 0];
            reg_a[1] = shared_a[(r + 1) * 16 + i + (r / 2 + 0) * 0];
            reg_a[2] = shared_a[(r + 2) * 16 + i + (r / 2 + 1) * 0];
            reg_a[3] = shared_a[(r + 3) * 16 + i + (r / 2 + 1) * 0];
            reg_b[0] = shared_b[(s + 0) * 16 + i + s / 2 * 0];
            reg_b[1] = shared_b[(s + 1) * 16 + i + s / 2 * 0];
            /*
            accum[0][0] += reg_a[0] * reg_b[0];
            accum[0][1] += reg_a[0] * reg_b[1];
            accum[1][0] += reg_a[1] * reg_b[0];
            accum[1][1] += reg_a[1] * reg_b[1];
            accum[2][0] += reg_a[2] * reg_b[0];
            accum[2][1] += reg_a[2] * reg_b[1];
            accum[3][0] += reg_a[3] * reg_b[0];
            accum[3][1] += reg_a[3] * reg_b[1];
            */
        }
    }

    // Reduce the accumulated `half2` array to `half` by summing the low and
    // high halves.
    half result[4][2];

    result[0][0] = __low2half(accum[0][0]) + __high2half(accum[0][0]);
    result[0][1] = __low2half(accum[0][1]) + __high2half(accum[0][1]);
    result[1][0] = __low2half(accum[1][0]) + __high2half(accum[1][0]);
    result[1][1] = __low2half(accum[1][1]) + __high2half(accum[1][1]);
    result[2][0] = __low2half(accum[2][0]) + __high2half(accum[2][0]);
    result[2][1] = __low2half(accum[2][1]) + __high2half(accum[2][1]);
    result[3][0] = __low2half(accum[3][0]) + __high2half(accum[3][0]);
    result[3][1] = __low2half(accum[3][1]) + __high2half(accum[3][1]);

    // Write the accumulated results to the output matrix.
    *(half2 *) &matrix_c[offset_c + (r + 0) * 32 + s] = *(half2 *) result[0];
    *(half2 *) &matrix_c[offset_c + (r + 1) * 32 + s] = *(half2 *) result[1];
    *(half2 *) &matrix_c[offset_c + (r + 2) * 32 + s] = *(half2 *) result[2];
    *(half2 *) &matrix_c[offset_c + (r + 3) * 32 + s] = *(half2 *) result[3];
}

/**
 * Compute half-precision sparse matrix multiplication with DSD mode.
 * 
 * It multiplies a sparse matrix with a dense matrix and create a new dense
 * matrix from the sparse layout.
 * 
 * Blocks               : (Total Batches, Sparse Rows, Sparse Columns)
 * Threads per Block    : 128
 */
/*
template <bool tr_a, bool tr_b>
__global__ void sparse_hmm_dsd_32x32x32_kernel(
    const half* __restrict__ matrix_a,
    const half* __restrict__ matrix_b,
          half* __restrict__ matrix_c,
    sparse_layout layout, int num_blocks, int size_m, int size_n, int size_k
) {
    __shared__ float shared_a[32 * 33], shared_b[32 * 33];
    float4 buffer_a, buffer_b;
    float accum[2][2] = { 0 };

    // Get an offset of each matrix and calculate mapping indices.
    int offset_a = blockIdx.x * num_blocks * 32 * 32;
    int offset_b = blockIdx.x * size_k * size_n;
    int offset_c = blockIdx.x * size_m * size_n;

    int m = blockIdx.y * 32;
    int n = blockIdx.z * 32;

    int p = threadIdx.x / 8;
    int q = threadIdx.x % 8 * 4;
    int r = threadIdx.x / 16 * 2;
    int s = threadIdx.x % 16 * 2;

    // Prefetch first tiles from matrices in global memory.
    auto iter = layout.begin(blockIdx.y);
    auto block = *iter;
    buffer_a = *(float4 *) &matrix_a[offset_a + (block.idx() * 32 * 32) + p * 32 + q];
    buffer_b = *(float4 *) &matrix_b[offset_b + (tr_b ? ((n + p) * size_k + (block.col() * 32 + q)) : ((block.col() * 32 + p) * size_n + (n + q)))];

    #pragma unroll 1
    while (iter.valid()) {
        // Commit the prefetched tiles to the shared memory storage.
        __syncthreads();
        shared_a[tr_a ? ((q + 0) * 33 + p) : (p * 33 + (q + 0))] = buffer_a.x;
        shared_a[tr_a ? ((q + 1) * 33 + p) : (p * 33 + (q + 1))] = buffer_a.y;
        shared_a[tr_a ? ((q + 2) * 33 + p) : (p * 33 + (q + 2))] = buffer_a.z;
        shared_a[tr_a ? ((q + 3) * 33 + p) : (p * 33 + (q + 3))] = buffer_a.w;
        shared_b[tr_b ? (p * 33 + (q + 0)) : ((q + 0) * 33 + p)] = buffer_b.x;
        shared_b[tr_b ? (p * 33 + (q + 1)) : ((q + 1) * 33 + p)] = buffer_b.y;
        shared_b[tr_b ? (p * 33 + (q + 2)) : ((q + 2) * 33 + p)] = buffer_b.z;
        shared_b[tr_b ? (p * 33 + (q + 3)) : ((q + 3) * 33 + p)] = buffer_b.w;
        __syncthreads();

        // Prefetch next tiles from matrices in global memory.
        iter.next();
        if (iter.valid()) {
            block = *iter;
            buffer_a = *(float4 *) &matrix_a[offset_a + (block.idx() * 32 * 32) + p * 32 + q];
            buffer_b = *(float4 *) &matrix_b[offset_b + (tr_b ? ((n + p) * size_k + (block.col() * 32 + q)) : ((block.col() * 32 + p) * size_n + (n + q)))];
        }

        // Accumulate the tiled matrix multiplications by loading sliced vectors
        // from the shared memory to local register file.
        #pragma unroll
        for (int i = 0; i < 32; ++ i) {
            float reg_a[2], reg_b[2];

            reg_a[0] = shared_a[(r + 0) * 33 + i];
            reg_a[1] = shared_a[(r + 1) * 33 + i];
            reg_b[0] = shared_b[(s + 0) * 33 + i];
            reg_b[1] = shared_b[(s + 1) * 33 + i];

            accum[0][0] += reg_a[0] * reg_b[0];
            accum[0][1] += reg_a[0] * reg_b[1];
            accum[1][0] += reg_a[1] * reg_b[0];
            accum[1][1] += reg_a[1] * reg_b[1];
        }
    }

    // Write the accumulated results to the output matrix.
    matrix_c[offset_c + (m + r + 0) * size_n + (n + s + 0)] = accum[0][0];
    matrix_c[offset_c + (m + r + 0) * size_n + (n + s + 1)] = accum[0][1];
    matrix_c[offset_c + (m + r + 1) * size_n + (n + s + 0)] = accum[1][0];
    matrix_c[offset_c + (m + r + 1) * size_n + (n + s + 1)] = accum[1][1];
}
*/

/**
 * Compute half-precision sparse matrix multiplication with DDS mode.
 * 
 * It multiplies a dense matrix with a sparse matrix and create a new dense
 * matrix from the sparse layout.
 * 
 * Blocks               : (Total Batches, Sparse Rows, Sparse Columns)
 * Threads per Block    : 128
 */
/*
template <bool tr_a, bool tr_b>
__global__ void sparse_hmm_dds_32x32x32_kernel(
    const half* __restrict__ matrix_a,
    const half* __restrict__ matrix_b,
          half* __restrict__ matrix_c,
    sparse_layout layout, int num_blocks, int size_m, int size_n, int size_k
) {
    __shared__ float shared_a[32 * 33], shared_b[32 * 33];
    float4 buffer_a, buffer_b;
    float accum[2][2] = { 0 };

    // Get an offset of each matrix and calculate mapping indices.
    int offset_a = blockIdx.x * size_m * size_k;
    int offset_b = blockIdx.x * num_blocks * 32 * 32;
    int offset_c = blockIdx.x * size_m * size_n;

    int m = blockIdx.y * 32;
    int n = blockIdx.z * 32;

    int p = threadIdx.x / 8;
    int q = threadIdx.x % 8 * 4;
    int r = threadIdx.x / 16 * 2;
    int s = threadIdx.x % 16 * 2;

    // Prefetch first tiles from matrices in global memory.
    auto iter = layout.begin(blockIdx.z);
    auto block = *iter;
    buffer_a = *(float4 *) &matrix_a[offset_a + (tr_a ? ((block.row() * 32 + p) * size_m + (m + q)) : ((m + p) * size_k + (block.row() * 32 + q)))];
    buffer_b = *(float4 *) &matrix_b[offset_b + (block.idx() * 32 * 32) + p * 32 + q];

    #pragma unroll 1
    while (iter.valid()) {
        // Commit the prefetched tiles to the shared memory storage.
        __syncthreads();
        shared_a[tr_a ? ((q + 0) * 33 + p) : (p * 33 + (q + 0))] = buffer_a.x;
        shared_a[tr_a ? ((q + 1) * 33 + p) : (p * 33 + (q + 1))] = buffer_a.y;
        shared_a[tr_a ? ((q + 2) * 33 + p) : (p * 33 + (q + 2))] = buffer_a.z;
        shared_a[tr_a ? ((q + 3) * 33 + p) : (p * 33 + (q + 3))] = buffer_a.w;
        shared_b[tr_b ? (p * 33 + (q + 0)) : ((q + 0) * 33 + p)] = buffer_b.x;
        shared_b[tr_b ? (p * 33 + (q + 1)) : ((q + 1) * 33 + p)] = buffer_b.y;
        shared_b[tr_b ? (p * 33 + (q + 2)) : ((q + 2) * 33 + p)] = buffer_b.z;
        shared_b[tr_b ? (p * 33 + (q + 3)) : ((q + 3) * 33 + p)] = buffer_b.w;
        __syncthreads();

        // Prefetch next tiles from matrices in global memory.
        iter.next();
        if (iter.valid()) {
            block = *iter;
            buffer_a = *(float4 *) &matrix_a[offset_a + (tr_a ? ((block.row() * 32 + p) * size_m + (m + q)) : ((m + p) * size_k + (block.row() * 32 + q)))];
            buffer_b = *(float4 *) &matrix_b[offset_b + (block.idx() * 32 * 32) + p * 32 + q];
        }

        // Accumulate the tiled matrix multiplications by loading sliced vectors
        // from the shared memory to local register file.
        #pragma unroll
        for (int i = 0; i < 32; ++ i) {
            float reg_a[2], reg_b[2];

            reg_a[0] = shared_a[(r + 0) * 33 + i];
            reg_a[1] = shared_a[(r + 1) * 33 + i];
            reg_b[0] = shared_b[(s + 0) * 33 + i];
            reg_b[1] = shared_b[(s + 1) * 33 + i];

            accum[0][0] += reg_a[0] * reg_b[0];
            accum[0][1] += reg_a[0] * reg_b[1];
            accum[1][0] += reg_a[1] * reg_b[0];
            accum[1][1] += reg_a[1] * reg_b[1];
        }
    }

    // Write the accumulated results to the output matrix.
    matrix_c[offset_c + (m + r + 0) * size_n + (n + s + 0)] = accum[0][0];
    matrix_c[offset_c + (m + r + 0) * size_n + (n + s + 1)] = accum[0][1];
    matrix_c[offset_c + (m + r + 1) * size_n + (n + s + 0)] = accum[1][0];
    matrix_c[offset_c + (m + r + 1) * size_n + (n + s + 1)] = accum[1][1];
}*/


void sparse_hmm_32x32x32_kernel_wrapper(
    const torch::Tensor &matrix_a,
    const torch::Tensor &matrix_b,
    const torch::Tensor &matrix_c,
    const std::string& mode, const layout_tensors& layout, int num_blocks,
    int num_batches, int size_m, int size_n, int size_k, bool tr_a, bool tr_b
) {
    dim3 blocks;
    if (mode == "sdd") blocks = dim3(num_blocks, num_batches);
    else blocks = dim3(num_batches, size_m / 32, size_n / 32);

    if      ( tr_a &&  tr_b && mode == "sdd") sparse_hmm_sdd_32x32x32_kernel< true,  true><<<blocks, 128>>>((half *) matrix_a.data_ptr<at::Half>(), (half *) matrix_b.data_ptr<at::Half>(), (half *) matrix_c.data_ptr<at::Half>(), layout, num_blocks, size_m, size_n, size_k);
    //else if ( tr_a &&  tr_b && mode == "dsd") sparse_hmm_dsd_32x32x32_kernel< true,  true><<<blocks, 128>>>((half *) matrix_a.data_ptr<at::Half>(), (half *) matrix_b.data_ptr<at::Half>(), (half *) matrix_c.data_ptr<at::Half>(), layout, num_blocks, size_m, size_n, size_k);
    //else if ( tr_a &&  tr_b && mode == "dds") sparse_hmm_dds_32x32x32_kernel< true,  true><<<blocks, 128>>>((half *) matrix_a.data_ptr<at::Half>(), (half *) matrix_b.data_ptr<at::Half>(), (half *) matrix_c.data_ptr<at::Half>(), layout, num_blocks, size_m, size_n, size_k);
    else if (!tr_a &&  tr_b && mode == "sdd") sparse_hmm_sdd_32x32x32_kernel<false,  true><<<blocks, 128>>>((half *) matrix_a.data_ptr<at::Half>(), (half *) matrix_b.data_ptr<at::Half>(), (half *) matrix_c.data_ptr<at::Half>(), layout, num_blocks, size_m, size_n, size_k);
    //else if (!tr_a &&  tr_b && mode == "dsd") sparse_hmm_dsd_32x32x32_kernel<false,  true><<<blocks, 128>>>((half *) matrix_a.data_ptr<at::Half>(), (half *) matrix_b.data_ptr<at::Half>(), (half *) matrix_c.data_ptr<at::Half>(), layout, num_blocks, size_m, size_n, size_k);
    //else if (!tr_a &&  tr_b && mode == "dds") sparse_hmm_dds_32x32x32_kernel<false,  true><<<blocks, 128>>>((half *) matrix_a.data_ptr<at::Half>(), (half *) matrix_b.data_ptr<at::Half>(), (half *) matrix_c.data_ptr<at::Half>(), layout, num_blocks, size_m, size_n, size_k);
    else if ( tr_a && !tr_b && mode == "ssd") sparse_hmm_sdd_32x32x32_kernel< true, false><<<blocks, 128>>>((half *) matrix_a.data_ptr<at::Half>(), (half *) matrix_b.data_ptr<at::Half>(), (half *) matrix_c.data_ptr<at::Half>(), layout, num_blocks, size_m, size_n, size_k);
    //else if ( tr_a && !tr_b && mode == "dsd") sparse_hmm_dsd_32x32x32_kernel< true, false><<<blocks, 128>>>((half *) matrix_a.data_ptr<at::Half>(), (half *) matrix_b.data_ptr<at::Half>(), (half *) matrix_c.data_ptr<at::Half>(), layout, num_blocks, size_m, size_n, size_k);
    //else if ( tr_a && !tr_b && mode == "dds") sparse_hmm_dds_32x32x32_kernel< true, false><<<blocks, 128>>>((half *) matrix_a.data_ptr<at::Half>(), (half *) matrix_b.data_ptr<at::Half>(), (half *) matrix_c.data_ptr<at::Half>(), layout, num_blocks, size_m, size_n, size_k);
    else if (!tr_a && !tr_b && mode == "sdd") sparse_hmm_sdd_32x32x32_kernel<false, false><<<blocks, 128>>>((half *) matrix_a.data_ptr<at::Half>(), (half *) matrix_b.data_ptr<at::Half>(), (half *) matrix_c.data_ptr<at::Half>(), layout, num_blocks, size_m, size_n, size_k);
    //else if (!tr_a && !tr_b && mode == "dsd") sparse_hmm_dsd_32x32x32_kernel<false, false><<<blocks, 128>>>((half *) matrix_a.data_ptr<at::Half>(), (half *) matrix_b.data_ptr<at::Half>(), (half *) matrix_c.data_ptr<at::Half>(), layout, num_blocks, size_m, size_n, size_k);
    //else if (!tr_a && !tr_b && mode == "dds") sparse_hmm_dds_32x32x32_kernel<false, false><<<blocks, 128>>>((half *) matrix_a.data_ptr<at::Half>(), (half *) matrix_b.data_ptr<at::Half>(), (half *) matrix_c.data_ptr<at::Half>(), layout, num_blocks, size_m, size_n, size_k);
}
