#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <>

#include <torch/extension.h>

#include "sparse_ops.h"
#include "layout_utils.cuh"
#include "tiling_utils.cuh"

#define LAUNCH_BOUNDS_TILE(T, ROWS, COLUMNS) \
    __launch_bounds__(tile<T, ROWS, COLUMNS>::THREADS, \
                      1024 / tile<T, ROWS, COLUMNS>::THREADS)


/**
 * Compute sparse matrix multiplication with SDD mode.
 * 
 * It computes a multiplication with a dense matrix with other dense matrix and
 * create a new sparse matrix through corresponding sparse layout.
 * 
 * Blocks               : (Sparse Blocks, Total Batches)
 * Threads per Block    : 256 for single precision,
 *                        128 for half precision
 */
__global__ void LAUNCH_BOUNDS_TILE(float, 32, 8) sparse_smm_sdd_32x32x8_kernel(
    const float* __restrict__ matrix_a,
    const float* __restrict__ matrix_b,
          float* __restrict__ matrix_c,
    sparse_layout layout, uint num_blocks,
    uint size_m, uint size_n, uint size_k,
    bool trans_a, bool trans_b
) {
    float accumulator[4] = { 0.0f, };

    uint lane_idx = threadIdx.x % warpSize;
    uint warp_idx = threadIdx.x / warpSize;

    auto block = layout.get(blockIdx.x);
    uint m = block.row() * 32;
    uint n = block.col() * 32;

    __shared__ tile<float, 32, 8>::storage storage_a, storage_b;

    tile<float, 32, 8>::loader loader_a(
        &matrix_a[blockIdx.y * size_m * size_k], storage_a,
        trans_a ? size_m : size_k, trans_a
    );
    tile<float, 32, 8>::loader loader_b(
        &matrix_b[blockIdx.y * size_k * size_n], storage_b,
        trans_b ? size_k : size_n, !trans_b
    );

    loader_a.prefetch(trans_a ? 0 : m, trans_a ? m : 0);
    loader_b.prefetch(trans_b ? n : 0, trans_b ? 0 : n);

    #pragma unroll 1
    for (uint k = 0; k < size_k; k += 8) {
        loader_a.commit(k / 8 % 2);
        loader_b.commit(k / 8 % 2);
        __syncthreads();

        if (k + 8 < size_k) {
            loader_a.prefetch(trans_a ? k + 8 : m, trans_a ? m : k + 8);
            loader_b.prefetch(trans_b ? n : k + 8, trans_b ? k + 8 : n);
        }

        #pragma unroll
        for (uint i = 0; i < 8; ++ i) {
            float local_a[4], local_b;

            #pragma unroll
            for (uint j = 0; j < 4; ++ j)
                local_a[j] = storage_a.get(k / 8 % 2, warp_idx * 4 + j, i);
            local_b = storage_b.get(k / 8 % 2, lane_idx, i);

            #pragma unroll
            for (uint j = 0; j < 4; ++ j)
                accumulator[j] += local_a[j] * local_b;
        }
    }

    #pragma unroll
    for (uint i = 0; i < 4; ++ i)
        matrix_c[(blockIdx.y * num_blocks + block.idx()) * 32 * 32
                 + (warp_idx * 4 + i) * 32 + lane_idx] = accumulator[i];
}


torch::Tensor sparse_matmul_single(
    torch::Tensor a, torch::Tensor b, const std::string& mode,
    const layout_tensors& row_layout, const layout_tensors& col_layout,
    bool trans_a, bool trans_b
) {
    // Select current sparse layout by the given sparse mode.
    auto layout = (mode == "sdd"
                   || mode == "dsd" && !trans_a
                   || mode == "dds" && trans_b) ? row_layout : col_layout;
    int64_t num_blocks = std::get<0>(layout).size(0) / 2;
    int64_t sparse_width = (std::get<1>(layout).size(0) - 1) * 32;

    // Get the dimension sizes from the tensors.
    int64_t size_m = mode.at(1) == 'd' ? a.size(trans_a ? -1 : -2)
                                       : sparse_width;
    int64_t size_n = mode.at(2) == 'd' ? b.size(trans_b ? -2 : -1)
                                       : sparse_width;
    int64_t size_k = mode.at(2) == 'd' ? b.size(trans_b ? -1 : -2)
                                       : a.size(trans_a ? -2 : -1);

    // Construct output tensor shape with preserving multiple batch dimensions.
    auto dense = mode.at(1) == 'd' ? a : b;
    auto shape = dense.sizes().slice(0, dense.dim() - 2).vec();

    if (mode.at(0) == 'd') shape.insert(shape.end(), { size_m, size_n });
    else shape.insert(shape.end(), { num_blocks, 32, 32 });

    // Merge the batch dimensions to one.
    a = a.flatten(0, mode.at(1) == 'd' ? -3 : -4);
    b = b.flatten(0, mode.at(2) == 'd' ? -3 : -4);

    int64_t num_batches = a.size(0);

    // Create an empty output tensor to store the multiplication result.
    torch::Tensor c;
    if (mode.at(0) == 'd') c = a.new_empty({ num_batches, size_m, size_n });
    else c = a.new_empty({ num_batches, num_blocks, 32, 32 });

    // Launch CUDA kernel with corresponding sparse mode and dimension sizes.
    dim3 blocks;
    if (mode == "sdd") blocks = dim3(num_blocks, num_batches);
    else blocks = dim3(num_batches,
                       (size_m + 32 - 1) / 32, (size_n + 32 - 1) / 32);

    auto kernel = mode == "sdd" ? sparse_smm_sdd_32x32x8_kernel :
                  mode == "dsd" ? sparse_smm_sdd_32x32x8_kernel :
                                  sparse_smm_sdd_32x32x8_kernel;
    kernel<<<blocks, 256>>>(
        a.data_ptr<float>(), b.data_ptr<float>(), c.data_ptr<float>(),
        layout, num_blocks, size_m, size_n, size_k, trans_a, trans_b
    );

    // Return the output tensor with the multiple batch dimensions.
    return c.reshape(shape);
}
